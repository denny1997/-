
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
using namespace std;
const int max_movie = 1683;
const int max_user = 944;
#define THREAD_NUM 256
#define BLOCK_NUM 32

__global__ static void test(float* rate,float* result)
{
	const int tid=threadIdx.x;
	const int bid=blockIdx.x;
	int i,k,j;
	float sum;
	for(i=bid*THREAD_NUM+tid;i<max_movie-1;i+=BLOCK_NUM*THREAD_NUM)
	{
		for(k=1;k<max_movie;k++)
		{
			if(k==i+1)
				continue;
			else
			{
				sum=0;
				//count=0;
				for(j=1;j<max_user;j++)
				{
					if((rate[j*max_movie+i+1]==-1)||(rate[j*max_movie+k]==-1))
						continue;
					else
					{
						sum=sum+(rate[j*max_movie+i+1]-rate[j*max_movie+k])*(rate[j*max_movie+i+1]-rate[j*max_movie+k]);
						//count++;
					}
				}
				result[(i+1)*max_movie+k]=sqrt(sum);
			}
		}


	}

}


int main()
{
	cout<<"start:"<<endl;
	FILE *fp = fopen("/home/3160102482/myhelloworld/ml-100k/u2.base","r");

	float a[max_movie*max_user];
	float* res;
	res=(float*)malloc(sizeof(float)*max_movie*max_movie);
	int movieid,userid,stamp,i,j;
	float rating;
	float* gpudata;
	float* result;
	float min;
	int minindex1,minindex2,minindex3,k;
	//cout<<"yes"<<endl;
	//cout<<"4";
	for(i=0;i<max_user;i++)
		for(j=0;j<max_movie;j++)
			a[i*max_movie+j]=-1;
	//cout<<"3";
	hipMalloc((void**)&gpudata,sizeof(float)*max_movie*max_user);
	hipMalloc((void**)&result,sizeof(float)*max_movie*max_movie);
	while(!feof(fp)){
		fscanf(fp,"%d%d%f%d",&userid, &movieid, &rating, &stamp);
		a[userid*max_movie+movieid]=rating;
	}
	fclose(fp);
	hipMemcpy(gpudata,a,sizeof(float)*max_movie*max_user,hipMemcpyHostToDevice);
	//cout<<"1";
	test<<<BLOCK_NUM,THREAD_NUM,0>>>(gpudata,result);
	hipMemcpy(res,result,sizeof(float)*max_movie*max_movie,hipMemcpyDeviceToHost);

	hipFree(gpudata);
	hipFree(result);
	fp = fopen("/home/3160102482/myhelloworld/ml-100k/u2.test","r");
	while(!feof(fp)){
			fscanf(fp,"%d%d%f%d",&userid, &movieid, &rating, &stamp);
			min=100;
			for(i=movieid*max_movie+1;i<(movieid+1)*max_movie;i++)
			{
				if(i==(movieid*max_movie+movieid))
					continue;
				else
				{
					k=i-movieid*max_movie;
					if((a[userid*max_movie+k]!=-1)&&(res[i]<min))
					{
						minindex1=k;
						min=res[i];
					}

				}
			}
			min=100;
			for(i=movieid*max_movie+1;i<(movieid+1)*max_movie;i++)
			{
				if((i==(movieid*max_movie+movieid))||(i==(movieid*max_movie+minindex1)))
					continue;
				else
				{
					k=i-movieid*max_movie;
					if((a[userid*max_movie+k]!=-1)&&(res[i]<min))
					{
						minindex2=k;
						min=res[i];
					}

				}
			}
			min=100;
			for(i=movieid*max_movie+1;i<(movieid+1)*max_movie;i++)
			{
				if((i==(movieid*max_movie+movieid))||(i==(movieid*max_movie+minindex1))||(i==(movieid*max_movie+minindex2)))
					continue;
				else
				{
					k=i-movieid*max_movie;
					if((a[userid*max_movie+k]!=-1)&&(res[i]<min))
					{
						minindex3=k;
						min=res[i];
					}

				}
			}
			cout<<"userid:"<<userid<<" ";
			cout<<"movieid:"<<movieid<<" "; 
			cout<<"estimate:"<<(float)(a[userid*max_movie+minindex1]+a[userid*max_movie+minindex2]+a[userid*max_movie+minindex3])/3<<" ";
			cout<<"real:"<<rating<<endl;
			cout<<endl;
		}
		fclose(fp);
	//cout<<res[1687];
}
